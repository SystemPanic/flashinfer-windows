#include "hip/hip_runtime.h"

#include <flashinfer/activation.cuh>
#include "pytorch_extension_utils.h"
#include <hip/hip_runtime.h>



using namespace flashinfer;


__device__ __forceinline__ float gelu_tanh(const float& val) {
  const float cdf =
      0.5f * (1.0f + math::tanh((0.7978845608028654f * (val + 0.044715f * val * val * val))));
  return val * cdf;
}


void gelu_tanh_and_mul(at::Tensor& out, at::Tensor& input, bool enable_pdl) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);

  const c10::cuda::OptionalCUDAGuard device_guard(out.device());
  auto stream = at::cuda::getCurrentCUDAStream();
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    hipLaunchConfig_t config;
    config.gridDim = num_tokens;
    config.blockDim = std::min(d / vec_size, 1024U);
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = enable_pdl;
    config.numAttrs = 1;
    config.attrs = attrs;

    auto kernel = flashinfer::activation::act_and_mul_kernel<c_type, gelu_tanh>;

    cudaLaunchKernelEx(&config, kernel, static_cast<c_type*>(out.data_ptr()),
                       static_cast<c_type*>(input.data_ptr()), d);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "Failed to launch kernel: ", hipGetErrorString(err));

    return true;
  });
}

TORCH_LIBRARY_FRAGMENT(TORCH_EXTENSION_NAME, m) {
  m.def("gelu_tanh_and_mul", gelu_tanh_and_mul);
}