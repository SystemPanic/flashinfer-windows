#include "hip/hip_runtime.h"
#include <flashinfer/attention/decode.cuh>
#include "batch_decode_config.inc"

using namespace flashinfer;

namespace flashinfer {

template hipError_t
BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, DefaultAttention<false, false, false, false>, Params>(
    Params params, hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);

};