#include "hip/hip_runtime.h"
#include <flashinfer/attention/decode.cuh>
#include "batch_decode_config.inc"

using namespace flashinfer;

namespace flashinfer {

template hipError_t
BatchDecodeWithPagedKVCacheDispatched<256, PosEncodingMode::kNone, DefaultAttention<false, true, true, false>, Params>(
    Params params, hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);

};