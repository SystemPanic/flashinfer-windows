#include "hip/hip_runtime.h"
#include <flashinfer/attention/decode.cuh>
#include "single_decode_config.inc"

using namespace flashinfer;

namespace flashinfer {

template hipError_t SingleDecodeWithKVCacheDispatched<
    64, PosEncodingMode::kNone, DefaultAttention<false, false, false, false>, Params>(
    Params params, hip_bfloat16* tmp,
    hipStream_t stream);

};