#include <flashinfer/attention/decode.cuh>
#include "single_decode_config.inc"

using namespace flashinfer;

namespace flashinfer {

template hipError_t SingleDecodeWithKVCacheDispatched<
    256, PosEncodingMode::kNone, DefaultAttention<false, true, true, false>, Params>(
    Params params, half* tmp,
    hipStream_t stream);

};