#include <flashinfer/attention/decode.cuh>
#include "batch_decode_config.inc"

using namespace flashinfer;

namespace flashinfer {

template hipError_t
BatchDecodeWithPagedKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<false, false, false, false>, Params>(
    Params params, half* tmp_v,
    float* tmp_s, hipStream_t stream);

};