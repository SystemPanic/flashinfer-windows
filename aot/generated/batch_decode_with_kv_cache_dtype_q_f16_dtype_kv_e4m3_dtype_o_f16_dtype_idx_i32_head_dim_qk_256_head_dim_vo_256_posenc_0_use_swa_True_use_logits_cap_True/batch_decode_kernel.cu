#include <flashinfer/attention/decode.cuh>
#include "batch_decode_config.inc"

using namespace flashinfer;

namespace flashinfer {

template hipError_t
BatchDecodeWithPagedKVCacheDispatched<256, PosEncodingMode::kNone, DefaultAttention<false, true, true, false>, Params>(
    Params params, half* tmp_v,
    float* tmp_s, hipStream_t stream);

};