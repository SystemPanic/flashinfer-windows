#include <flashinfer/attention/prefill.cuh>
#include "single_prefill_config.inc"

using namespace flashinfer;

namespace flashinfer {

constexpr auto use_custom_mask = MaskMode::kNone == MaskMode::kCustom;

template hipError_t SinglePrefillWithKVCacheDispatched<
    128, 128, PosEncodingMode::kNone, false, MaskMode::kNone, DefaultAttention<use_custom_mask, false, false, false>, Params>(
    Params params, half* tmp,
    hipStream_t stream);

};