#include "hip/hip_runtime.h"
#include <flashinfer/attention/decode.cuh>
#include "single_decode_config.inc"

using namespace flashinfer;

namespace flashinfer {

template hipError_t SingleDecodeWithKVCacheDispatched<
    256, PosEncodingMode::kNone, DefaultAttention<false, true, true, false>, Params>(
    Params params, hip_bfloat16* tmp,
    hipStream_t stream);

};