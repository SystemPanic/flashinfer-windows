#include "hip/hip_runtime.h"
#include <flashinfer/attention/prefill.cuh>
#include "single_prefill_config.inc"

using namespace flashinfer;

namespace flashinfer {

constexpr auto use_custom_mask = MaskMode::kCustom == MaskMode::kCustom;

template hipError_t SinglePrefillWithKVCacheDispatched<
    128, 128, PosEncodingMode::kNone, false, MaskMode::kCustom, DefaultAttention<use_custom_mask, false, false, false>, Params>(
    Params params, hip_bfloat16* tmp,
    hipStream_t stream);

};